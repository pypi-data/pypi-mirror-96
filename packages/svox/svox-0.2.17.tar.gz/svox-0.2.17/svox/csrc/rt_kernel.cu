#include "hip/hip_runtime.h"
/*
 * Copyright Alex Yu 2021
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice,
 * this list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include <cstdint>
#include "common.cuh"

namespace {
int cuda_n_threads = -1;
__host__ void auto_cuda_threads() {
    if (~cuda_n_threads) return;
    hipDeviceProp_t dev_prop;
    hipGetDeviceProperties(&dev_prop, 0);
    const int n_cores = get_sp_cores(dev_prop);
    // Optimize number of CUDA threads per block
    if (n_cores < 2048) {
        cuda_n_threads = 256;
    } if (n_cores < 8192) {
        cuda_n_threads = 512;
    } else {
        cuda_n_threads = 1024;
    }
}

namespace device {
// SH Coefficients from https://github.com/google/spherical-harmonics
__device__ __constant__ const float C0 = 0.28209479177387814;
__device__ __constant__ const float C1 = 0.4886025119029199;
__device__ __constant__ const float C2[] = {
    1.0925484305920792,
    -1.0925484305920792,
    0.31539156525252005,
    -1.0925484305920792,
    0.5462742152960396
};

__device__ __constant__ const float C3[] = {
    -0.5900435899266435,
    2.890611442640554,
    -0.4570457994644658,
    0.3731763325901154,
    -0.4570457994644658,
    1.445305721320277,
    -0.5900435899266435
};

__device__ __constant__ const float C4[] = {
    2.5033429417967046,
    -1.7701307697799304,
    0.9461746957575601,
    -0.6690465435572892,
    0.10578554691520431,
    -0.6690465435572892,
    0.47308734787878004,
    -1.7701307697799304,
    0.6258357354491761,
};

template<typename scalar_t>
__host__ __device__ __inline__ static scalar_t _norm(
                scalar_t* dir) {
    return sqrtf(dir[0] * dir[0] + dir[1] * dir[1] + dir[2] * dir[2]);
}

template<typename scalar_t>
__host__ __device__ __inline__ static void _normalize(
                scalar_t* dir) {
    scalar_t norm = _norm(dir);
    dir[0] /= norm; dir[1] /= norm; dir[2] /= norm;
}


// Calculate SH basis functions of given order, for given view directions
template <typename scalar_t>
__device__ __inline__ void _precalc_sh(
    const int order,
    const scalar_t* __restrict__ dir,
    scalar_t* __restrict__ out_mult) {

    out_mult[0] = C0;
    const float x = dir[0], y = dir[1], z = dir[2];
    const float xx = x * x, yy = y * y, zz = z * z;
    const float xy = x * y, yz = y * z, xz = x * z;
    switch (order) {
        case 4:
            out_mult[16] = C4[0] * xy * (xx - yy);
            out_mult[17] = C4[1] * yz * (3 * xx - yy);
            out_mult[18] = C4[2] * xy * (7 * zz - 1.f);
            out_mult[19] = C4[3] * yz * (7 * zz - 3.f);
            out_mult[20] = C4[4] * (zz * (35 * zz - 30) + 3);
            out_mult[21] = C4[5] * xz * (7 * zz - 3);
            out_mult[22] = C4[6] * (xx - yy) * (7 * zz - 1.f);
            out_mult[23] = C4[7] * xz * (xx - 3 * yy);
            out_mult[24] = C4[8] * (xx * (xx - 3 * yy) - yy * (3 * xx - yy));
            [[fallthrough]];
        case 3:
            out_mult[9] = C3[0] * y * (3 * xx - yy);
            out_mult[10] = C3[1] * xy * z;
            out_mult[11] = C3[2] * y * (4 * zz - xx - yy);
            out_mult[12] = C3[3] * z * (2 * zz - 3 * xx - 3 * yy);
            out_mult[13] = C3[4] * x * (4 * zz - xx - yy);
            out_mult[14] = C3[5] * z * (xx - yy);
            out_mult[15] = C3[6] * x * (xx - 3 * yy);
            [[fallthrough]];
        case 2:
            out_mult[4] = C2[0] * xy;
            out_mult[5] = C2[1] * yz;
            out_mult[6] = C2[2] * (2.0 * zz - xx - yy);
            out_mult[7] = C2[3] * xz;
            out_mult[8] = C2[4] * (xx - yy);
            [[fallthrough]];
        case 1:
            out_mult[1] = -C1 * y;
            out_mult[2] = C1 * z;
            out_mult[3] = -C1 * x;
    }
}

template <typename scalar_t>
__device__ __inline__ scalar_t _get_delta_scale(
    const scalar_t* __restrict__ scaling,
    scalar_t* __restrict__ dir) {
    dir[0] *= scaling[0];
    dir[1] *= scaling[1];
    dir[2] *= scaling[2];
    scalar_t delta_scale = 1.f / _norm(dir);
    dir[0] *= delta_scale;
    dir[1] *= delta_scale;
    dir[2] *= delta_scale;
    return delta_scale;
}

template <typename scalar_t>
__device__ __inline__ void _dda_unit(
        const scalar_t* __restrict__ cen,
        const scalar_t* __restrict__ invdir,
        scalar_t* __restrict__ tmin,
        scalar_t* __restrict__ tmax) {
    // Perform DDA for 1 iteration on a unit cube
    scalar_t t1, t2;
    *tmin = 0.0f;
    *tmax = 1e9f;
#pragma unroll
    for (int i = 0; i < 3; ++i) {
        t1 = - cen[i] * invdir[i];
        t2 = t1 +  invdir[i];
        *tmin = max(*tmin, min(t1, t2));
        *tmax = min(*tmax, max(t1, t2));
    }
}


template <typename scalar_t>
__device__ __inline__ void trace_ray(
    const torch::PackedTensorAccessor32<torch::Half, 5, torch::RestrictPtrTraits>
        data,
    const torch::PackedTensorAccessor32<int32_t, 4, torch::RestrictPtrTraits>
        child,
        const scalar_t* __restrict__ origin,
        const scalar_t* __restrict__ dir,
        const scalar_t* __restrict__ vdir,
        scalar_t step_size,
        scalar_t background_brightness,
        int sh_order,
        scalar_t delta_scale,
        scalar_t sigma_thresh,
        scalar_t stop_thresh,
        torch::TensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, int32_t> out,
        scalar_t* __restrict__ weight_accum = nullptr) {

    scalar_t tmin, tmax;
    scalar_t invdir[3];
    const int tree_N = child.size(1);
    const int data_dim = data.size(4);
    const int out_data_dim = out.size(0);

#pragma unroll
    for (int i = 0; i < 3; ++i) {
        invdir[i] = 1.0 / (dir[i] + 1e-9);
    }
    _dda_unit(origin, invdir, &tmin, &tmax);

    if (tmax < 0 || tmin > tmax) {
        // Ray doesn't hit box
        for (int j = 0; j < out_data_dim; ++j) {
            out[j] = background_brightness;
        }
        return;
    } else {
        for (int j = 0; j < out_data_dim; ++j) {
            out[j] = 0.f;
        }
        scalar_t pos[3];
        scalar_t sh_mult[25];
        if (sh_order >= 0) {
            _precalc_sh<scalar_t>(sh_order, vdir, sh_mult);
        }

        scalar_t light_intensity = 1.f;
        scalar_t t = tmin;
        const int n_coe = (sh_order + 1) * (sh_order + 1);
        scalar_t cube_sz;
        while (t < tmax) {
            for (int j = 0; j < 3; ++j) {
                pos[j] = origin[j] + t * dir[j];
            }

            int32_t node_id;
            torch::Half* tree_val = query_single_from_root<scalar_t>(data, child,
                        pos, &cube_sz, &node_id);

            scalar_t att;
            scalar_t subcube_tmin, subcube_tmax;
            _dda_unit(pos, invdir, &subcube_tmin, &subcube_tmax);

            const scalar_t t_subcube = (subcube_tmax - subcube_tmin) / cube_sz;
            const scalar_t delta_t = t_subcube + step_size;
            const scalar_t sigma = __half2float(tree_val[data_dim - 1]);
            if (sigma > sigma_thresh) {
                att = expf(-delta_t * delta_scale * sigma);
                const scalar_t weight = light_intensity * (1.f - att);

                if (sh_order >= 0) {
                    for (int t = 0; t < out_data_dim; ++ t) {
                        int off = t * n_coe;
                        scalar_t tmp = 0.0;
                        for (int i = 0; i < n_coe; ++i) {
                            tmp += sh_mult[i] * __half2float(tree_val[off + i]);
                        }
                        out[t] += weight / (1.0 + expf(-tmp));
                    }
                } else {
                    for (int j = 0; j < out_data_dim; ++j) {
                        out[j] += weight / (1.0 + expf(-__half2float(tree_val[j])));
                    }
                }
                light_intensity *= att;

                if (weight_accum != nullptr) {
                    weight_accum[node_id] += weight;
                }

                if (light_intensity <= stop_thresh) {
                    // Full opacity, stop
                    scalar_t scale = 1.0 / (1.0 - light_intensity);
                    out[0] *= scale; out[1] *= scale; out[2] *= scale;
                    return;
                }
            }
            t += delta_t;
        }
        for (int j = 0; j < out_data_dim; ++j) {
            out[j] += light_intensity * background_brightness;
        }
    }
}

template <typename scalar_t>
__device__ __inline__ void trace_ray_backward(
    const torch::PackedTensorAccessor32<torch::Half, 5, torch::RestrictPtrTraits>
        data,
    const torch::PackedTensorAccessor32<int32_t, 4, torch::RestrictPtrTraits>
        child,
    const torch::TensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, int32_t>
        grad_out,
        const scalar_t* __restrict__ origin,
        const scalar_t* __restrict__ dir,
        const scalar_t* __restrict__ vdir,
        scalar_t step_size,
        scalar_t background_brightness,
        int sh_order,
        scalar_t delta_scale,
    torch::PackedTensorAccessor32<scalar_t, 5, torch::RestrictPtrTraits>
        grad_data) {

    scalar_t tmin, tmax;
    scalar_t invdir[3];
    const int tree_N = child.size(1);
    const int data_dim = data.size(4);
    const int out_data_dim = grad_out.size(0);

#pragma unroll
    for (int i = 0; i < 3; ++i) {
        invdir[i] = 1.0 / (dir[i] + 1e-9);
    }
    _dda_unit(origin, invdir, &tmin, &tmax);

    if (tmax < 0 || tmin > tmax) {
        // Ray doesn't hit box
        return;
    } else {
        scalar_t pos[3];
        scalar_t sh_mult[16];
        if (sh_order >= 0) {
            _precalc_sh<scalar_t>(sh_order, vdir, sh_mult);
        }
        const int n_coe = (sh_order + 1) * (sh_order + 1);

        scalar_t accum = 0.0;
        // PASS 1
        {
            scalar_t light_intensity = 1.f, t = tmin, cube_sz;
            while (t < tmax) {
                for (int j = 0; j < 3; ++j) pos[j] = origin[j] + t * dir[j];

                int32_t _node_id;
                const torch::Half* tree_val = query_single_from_root<scalar_t>(data, child,
                        pos, &cube_sz, &_node_id);
                // Reuse offset on gradient
                const int curr_leaf_offset = tree_val - data.data();
                scalar_t* grad_tree_val = grad_data.data() + curr_leaf_offset;

                scalar_t att;
                scalar_t subcube_tmin, subcube_tmax;
                _dda_unit(pos, invdir, &subcube_tmin, &subcube_tmax);

                const scalar_t t_subcube = (subcube_tmax - subcube_tmin) / cube_sz;
                const scalar_t delta_t = t_subcube + step_size;
                const scalar_t sigma = __half2float(tree_val[data_dim - 1]);
                if (sigma > 0.0) {
                    att = expf(-delta_t * sigma * delta_scale);
                    const scalar_t weight = light_intensity * (1.f - att);

                    scalar_t total_color = 0.f;
                    if (sh_order >= 0) {
                        for (int t = 0; t < out_data_dim; ++ t) {
                            int off = t * n_coe;
                            scalar_t tmp = 0.0;
                            for (int i = 0; i < n_coe; ++i) {
                                tmp += sh_mult[i] * __half2float(tree_val[off + i]);
                            }
                            const scalar_t sigmoid = 1.0 / (1.0 + expf(-tmp));
                            const scalar_t grad_sigmoid = sigmoid * (1.0 - sigmoid);
                            for (int i = 0; i < n_coe; ++i) {
                                const scalar_t toadd = weight * sh_mult[i] *
                                    grad_sigmoid * grad_out[t];
                                atomicAdd(&grad_tree_val[off + i],
                                        toadd);
                            }
                            total_color += sigmoid * grad_out[t];
                        }
                    } else {
                        for (int j = 0; j < out_data_dim; ++j) {
                            const scalar_t sigmoid = 1.0 / (1.0 + expf(
                                        -__half2float(tree_val[j])));
                            const scalar_t toadd = weight * sigmoid * (1.f - sigmoid) * grad_out[j];
                            atomicAdd(&grad_tree_val[j], toadd);
                            total_color += sigmoid * grad_out[j];
                        }
                    }
                    light_intensity *= att;
                    accum += weight * total_color;
                }
                t += delta_t;
            }
            scalar_t total_grad = 0.f;
            for (int j = 0; j < out_data_dim; ++j)
                total_grad += grad_out[j];
            accum += light_intensity * background_brightness * total_grad;
        }
        // PASS 2
        {
            // scalar_t accum_lo = 0.0;
            scalar_t light_intensity = 1.f, t = tmin, cube_sz;
            while (t < tmax) {
                for (int j = 0; j < 3; ++j) pos[j] = origin[j] + t * dir[j];
                int32_t _node_id;
                const torch::Half* tree_val = query_single_from_root<scalar_t>(data, child,
                        pos, &cube_sz, &_node_id);
                // Reuse offset on gradient
                const int curr_leaf_offset = tree_val - data.data();
                scalar_t* grad_tree_val = grad_data.data() + curr_leaf_offset;

                scalar_t att;
                scalar_t subcube_tmin, subcube_tmax;
                _dda_unit(pos, invdir, &subcube_tmin, &subcube_tmax);

                const scalar_t t_subcube = (subcube_tmax - subcube_tmin) / cube_sz;
                const scalar_t delta_t = t_subcube + step_size;
                const scalar_t sigma = __half2float(tree_val[data_dim - 1]);
                if (sigma > 0.0) {
                    att = expf(-delta_t * sigma * delta_scale);
                    const scalar_t weight = light_intensity * (1.f - att);

                    scalar_t total_color = 0.f;
                    if (sh_order >= 0) {
                        for (int t = 0; t < out_data_dim; ++ t) {
                            int off = t * n_coe;
                            scalar_t tmp = 0.0;
                            for (int i = 0; i < n_coe; ++i) {
                                tmp += sh_mult[i] * __half2float(tree_val[off + i]);
                            }
                            total_color += 1.0 / (1.0 + expf(-tmp)) * grad_out[t];
                        }
                    } else {
                        for (int j = 0; j < out_data_dim; ++j) {
                            total_color += 1.0 / (1.0 + expf(-__half2float(tree_val[j]))) * grad_out[j];
                        }
                    }
                    light_intensity *= att;
                    accum -= weight * total_color;
                    atomicAdd(
                            &grad_tree_val[out_data_dim],
                            delta_t * delta_scale * (
                                total_color * light_intensity - accum)
                            );
                }
                t += delta_t;
            }
        }
    }
}

template <typename scalar_t>
__global__ void render_ray_kernel(
    const torch::PackedTensorAccessor32<torch::Half, 5, torch::RestrictPtrTraits>
        data,
    const torch::PackedTensorAccessor32<int32_t, 4, torch::RestrictPtrTraits>
        child,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits>
        origins,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits>
        dirs,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits>
        vdirs,
    scalar_t step_size,
    scalar_t background_brightness,
    int sh_order,
    scalar_t sigma_thresh,
    scalar_t stop_thresh,
    const scalar_t* __restrict__ offset,
    const scalar_t* __restrict__ scaling,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits>
        out,
    scalar_t* __restrict__ weight_accum) {
    CUDA_GET_THREAD_ID(tid, origins.size(0));
    scalar_t origin[3] = {origins[tid][0], origins[tid][1], origins[tid][2]};
    transform_coord<scalar_t>(origin, offset, scaling);
    scalar_t dir[3] = {dirs[tid][0], dirs[tid][1], dirs[tid][2]};
    const scalar_t delta_scale = _get_delta_scale(scaling, dir);

    trace_ray<scalar_t>(
        data, child,
        origin,
        dir,
        &vdirs[tid][0],
        step_size,
        background_brightness,
        sh_order,
        delta_scale,
        sigma_thresh,
        stop_thresh,
        out[tid],
        weight_accum);
}


template <typename scalar_t>
__global__ void render_ray_backward_kernel(
    const torch::PackedTensorAccessor32<torch::Half, 5, torch::RestrictPtrTraits>
        data,
    const torch::PackedTensorAccessor32<int32_t, 4, torch::RestrictPtrTraits>
        child,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits>
        grad_out,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits>
        origins,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits>
        dirs,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits>
        vdirs,
    scalar_t step_size,
    scalar_t background_brightness,
    int sh_order,
    const scalar_t* __restrict__ offset,
    const scalar_t* __restrict__ scaling,
    torch::PackedTensorAccessor32<scalar_t, 5, torch::RestrictPtrTraits>
        grad_data
        ) {
    CUDA_GET_THREAD_ID(tid, origins.size(0));
    scalar_t origin[3] = {origins[tid][0], origins[tid][1], origins[tid][2]};
    transform_coord<scalar_t>(origin, offset, scaling);
    scalar_t dir[3] = {dirs[tid][0], dirs[tid][1], dirs[tid][2]};
    const scalar_t delta_scale = _get_delta_scale(scaling, dir);
    trace_ray_backward<scalar_t>(
        data, child,
        grad_out[tid],
        origin,
        dir,
        &vdirs[tid][0],
        step_size,
        background_brightness,
        sh_order,
        delta_scale,
        grad_data);
}

template <typename scalar_t>
__device__ __inline__ void cam2world_ray(
    int ix, int iy,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits>
        c2w,
    scalar_t* dir,
    scalar_t* origin,
    scalar_t fx, scalar_t fy,
    int width, int height) {
    scalar_t x = (ix - 0.5 * width) / fx;
    scalar_t y = -(iy - 0.5 * height) / fy;
    scalar_t z = sqrtf(x * x + y * y + 1.0);
    x /= z; y /= z; z = -1.0f / z;
    dir[0] = c2w[0][0] * x + c2w[0][1] * y + c2w[0][2] * z;
    dir[1] = c2w[1][0] * x + c2w[1][1] * y + c2w[1][2] * z;
    dir[2] = c2w[2][0] * x + c2w[2][1] * y + c2w[2][2] * z;
    origin[0] = c2w[0][3]; origin[1] = c2w[1][3]; origin[2] = c2w[2][3];
}


template <typename scalar_t>
__host__ __device__ __inline__ static void world2ndc(
        int ndc_width, int ndc_height, scalar_t ndc_focal,
        scalar_t* __restrict__ dir,
        scalar_t* __restrict__ cen, scalar_t near = 1.f) {
    scalar_t t = -(near + cen[2]) / dir[2];
    for (int i = 0; i < 3; ++i) {
        cen[i] = cen[i] + t * dir[i];
    }

    dir[0] = -((2 * ndc_focal) / ndc_width) * (dir[0] / dir[2] - cen[0] / cen[2]);
    dir[1] = -((2 * ndc_focal) / ndc_height) * (dir[1] / dir[2] - cen[1] / cen[2]);
    dir[2] = -2 * near / cen[2];

    cen[0] = -((2 * ndc_focal) / ndc_width) * (cen[0] / cen[2]);
    cen[1] = -((2 * ndc_focal) / ndc_height) * (cen[1] / cen[2]);
    cen[2] = 1 + 2 * near / cen[2];

    _normalize(dir);
}


template <typename scalar_t>
__global__ void render_image_kernel(
    const torch::PackedTensorAccessor32<torch::Half, 5, torch::RestrictPtrTraits>
        data,
    const torch::PackedTensorAccessor32<int32_t, 4, torch::RestrictPtrTraits>
        child,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits>
        c2w,
    scalar_t step_size,
    scalar_t background_brightness,
    int sh_order,
    scalar_t sigma_thresh,
    scalar_t stop_thresh,
    scalar_t fx,
    scalar_t fy,
    int width,
    int height,
    scalar_t ndc_focal,
    int ndc_width,
    int ndc_height,
    const scalar_t* __restrict__ offset,
    const scalar_t* __restrict__ scaling,
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits>
        out,
    scalar_t* __restrict__ weight_accum) {
    CUDA_GET_THREAD_ID(tid, width * height);
    int iy = tid / width, ix = tid % width;
    scalar_t dir[3], origin[3];
    cam2world_ray(ix, iy, c2w, dir, origin, fx, fy, width, height);
    scalar_t vdir[3] = {dir[0], dir[1], dir[2]};
    if (ndc_width > 1) {
        world2ndc(ndc_width, ndc_height, ndc_focal, dir, origin);
    }

    transform_coord<scalar_t>(origin, offset, scaling);
    const scalar_t delta_scale = _get_delta_scale(scaling, dir);
    trace_ray<scalar_t>(
        data, child,
        origin,
        dir,
        vdir,
        step_size,
        background_brightness,
        sh_order,
        delta_scale,
        sigma_thresh,
        stop_thresh,
        out[iy][ix],
        weight_accum);
}

template <typename scalar_t>
__global__ void render_image_backward_kernel(
    const torch::PackedTensorAccessor32<torch::Half, 5, torch::RestrictPtrTraits>
        data,
    const torch::PackedTensorAccessor32<int32_t, 4, torch::RestrictPtrTraits>
        child,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits>
        c2w,
    const torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits>
        grad_out,
    scalar_t step_size,
    scalar_t background_brightness,
    int sh_order,
    scalar_t fx,
    scalar_t fy,
    int width,
    int height,
    scalar_t ndc_focal,
    int ndc_width,
    int ndc_height,
    const scalar_t* __restrict__ offset,
    const scalar_t* __restrict__ scaling,
    torch::PackedTensorAccessor32<scalar_t, 5, torch::RestrictPtrTraits>
        grad_data
        ) {
    CUDA_GET_THREAD_ID(tid, width * height);
    int iy = tid / width, ix = tid % width;
    scalar_t dir[3], origin[3];
    cam2world_ray(ix, iy, c2w, dir, origin, fx, fy, width, height);
    scalar_t vdir[3] = {dir[0], dir[1], dir[2]};
    if (ndc_width > 1) {
        world2ndc(ndc_width, ndc_height, ndc_focal, dir, origin);
    }

    transform_coord<scalar_t>(origin, offset, scaling);
    const scalar_t delta_scale = _get_delta_scale(scaling, dir);
    trace_ray_backward<scalar_t>(
        data, child,
        grad_out[iy][ix],
        origin,
        dir,
        vdir,
        step_size,
        background_brightness,
        sh_order,
        delta_scale,
        grad_data);
}

}  // namespace device


// Compute RGB output dimension from input dimension & SH order
__host__ int get_out_data_dim(int sh_order, int in_data_dim) {
    int out_data_dim;
    if (sh_order >= 0) {
        const int n_coe = (sh_order + 1) * (sh_order + 1);
        out_data_dim = (in_data_dim - 1) / n_coe;
    } else {
        out_data_dim = in_data_dim - 1;
    }
    return out_data_dim;
}

}  // namespace

torch::Tensor _volume_render_cuda(torch::Tensor data, torch::Tensor child,
                            torch::Tensor origins, torch::Tensor dirs,
                            torch::Tensor vdirs, torch::Tensor offset,
                            torch::Tensor scaling, float step_size,
                            float background_brightness,
                            int sh_order, bool fast, at::Tensor weight_accum) {
    const auto Q = origins.size(0);

    const float sigma_thresh = fast ? 1e-2f : 0.f;
    const float stop_thresh = fast ? 1e-2f : 0.f;

    auto_cuda_threads();
    const int blocks = CUDA_N_BLOCKS_NEEDED(Q, cuda_n_threads);
    int out_data_dim = get_out_data_dim(sh_order, data.size(4));
    torch::Tensor result = torch::zeros({Q, out_data_dim}, origins.options());
    AT_DISPATCH_FLOATING_TYPES(origins.type(), volume_render, [&] {
            device::render_ray_kernel<scalar_t><<<blocks, cuda_n_threads>>>(
                data.packed_accessor32<torch::Half, 5, torch::RestrictPtrTraits>(),
                child.packed_accessor32<int32_t, 4, torch::RestrictPtrTraits>(),
                origins.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                dirs.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                vdirs.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                step_size,
                background_brightness,
                sh_order,
                sigma_thresh,
                stop_thresh,
                offset.data<scalar_t>(),
                scaling.data<scalar_t>(),
                result.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                weight_accum.numel() > 0 ?  weight_accum.data<scalar_t>() : nullptr);
    });
    CUDA_CHECK_ERRORS;
    return result;
}

torch::Tensor _volume_render_image_cuda(
    torch::Tensor data, torch::Tensor child, torch::Tensor offset,
    torch::Tensor scaling, torch::Tensor c2w, float fx, float fy, int width,
    int height, float step_size,
    float background_brightness, int sh_order, int ndc_width, int ndc_height,
    float ndc_focal, bool fast, at::Tensor weight_accum) {
    const size_t Q = size_t(width) * height;

    const float sigma_thresh = fast ? 1e-2f : 0.f;
    const float stop_thresh = fast ? 1e-2f : 0.f;

    auto_cuda_threads();
    const int blocks = CUDA_N_BLOCKS_NEEDED(Q, cuda_n_threads);
    int out_data_dim = get_out_data_dim(sh_order, data.size(4));
    torch::Tensor result = torch::zeros({height, width, out_data_dim}, c2w.options());

    AT_DISPATCH_FLOATING_TYPES(c2w.type(), volume_render_image, [&] {
            device::render_image_kernel<scalar_t><<<blocks, cuda_n_threads>>>(
                data.packed_accessor32<torch::Half, 5, torch::RestrictPtrTraits>(),
                child.packed_accessor32<int32_t, 4, torch::RestrictPtrTraits>(),
                c2w.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                step_size,
                background_brightness,
                sh_order,
                sigma_thresh,
                stop_thresh,
                fx,
                fy,
                width,
                height,
                ndc_focal,
                ndc_width,
                ndc_height,
                offset.data<scalar_t>(),
                scaling.data<scalar_t>(),
                result.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
                weight_accum.numel() > 0 ?  weight_accum.data<scalar_t>() : nullptr);
    });
    CUDA_CHECK_ERRORS;
    return result;
}

torch::Tensor _volume_render_backward_cuda(
    torch::Tensor data, torch::Tensor child, torch::Tensor grad_output,
    torch::Tensor origins, torch::Tensor dirs, torch::Tensor vdirs,
    torch::Tensor offset, torch::Tensor scaling, float step_size,
    float background_brightness, int sh_order) {
    const int Q = origins.size(0);

    auto_cuda_threads();
    const int blocks = CUDA_N_BLOCKS_NEEDED(Q, cuda_n_threads);
    int out_data_dim = get_out_data_dim(sh_order, data.size(4));
    torch::Tensor result = torch::zeros(data.sizes(), grad_output.options());
    AT_DISPATCH_FLOATING_TYPES(origins.type(), volume_render_backward, [&] {
            device::render_ray_backward_kernel<scalar_t><<<blocks, cuda_n_threads>>>(
                data.packed_accessor32<torch::Half, 5, torch::RestrictPtrTraits>(),
                child.packed_accessor32<int32_t, 4, torch::RestrictPtrTraits>(),
                grad_output.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                origins.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                dirs.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                vdirs.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                step_size,
                background_brightness,
                sh_order,
                offset.data<scalar_t>(),
                scaling.data<scalar_t>(),
                result.packed_accessor32<scalar_t, 5, torch::RestrictPtrTraits>());
    });
    CUDA_CHECK_ERRORS;
    return result;
}

torch::Tensor _volume_render_image_backward_cuda(
    torch::Tensor data, torch::Tensor child, torch::Tensor grad_output,
    torch::Tensor offset, torch::Tensor scaling, torch::Tensor c2w, float fx,
    float fy, int width, int height, float step_size,
    float background_brightness, int sh_order, int ndc_width, int ndc_height,
    float ndc_focal) {
    const size_t Q = size_t(width) * height;

    auto_cuda_threads();
    const int blocks = CUDA_N_BLOCKS_NEEDED(Q, cuda_n_threads);
    int out_data_dim = get_out_data_dim(sh_order, data.size(4));
    torch::Tensor result = torch::zeros(data.sizes(), grad_output.options());

    AT_DISPATCH_FLOATING_TYPES(grad_output.type(), volume_render_image_backward, [&] {
            device::render_image_backward_kernel<scalar_t><<<blocks, cuda_n_threads>>>(
                data.packed_accessor32<torch::Half, 5, torch::RestrictPtrTraits>(),
                child.packed_accessor32<int32_t, 4, torch::RestrictPtrTraits>(),
                c2w.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                grad_output.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
                step_size,
                background_brightness,
                sh_order,
                fx,
                fy,
                width,
                height,
                ndc_focal,
                ndc_width,
                ndc_height,
                offset.data<scalar_t>(),
                scaling.data<scalar_t>(),
                result.packed_accessor32<scalar_t, 5, torch::RestrictPtrTraits>());
    });
    CUDA_CHECK_ERRORS;
    return result;
}
