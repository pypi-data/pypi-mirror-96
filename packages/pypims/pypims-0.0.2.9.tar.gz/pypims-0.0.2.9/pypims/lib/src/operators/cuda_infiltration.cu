#include "hip/hip_runtime.h"
// ======================================================================================
// Name                :    High-Performance Integrated Modelling System
// Description         :    This code pack provides a generic framework for developing 
//                          Geophysical CFD software. Legacy name: GeoClasses
// ======================================================================================
// Version             :    1.0.1 
// Author              :    Xilin Xia
// Create Time         :    2014/10/04
// Update Time         :    2020/04/26
// ======================================================================================
// LICENCE: GPLv3 
// ======================================================================================

/*!
\file cuda_infiltration.cu
\brief Source file for friction operator

\version 0.1
\author xilin xia

*/

#include "cuda_infiltration.h"
#include "cuda_kernel_launch_parameters.h"

namespace GC{

  namespace fv{

    __global__ void cuInfiltrationGreenAmptKernel(Scalar* h, Scalar* hydraulic_conductivity, Scalar* capillary_head, Scalar* water_content_diff, Scalar* culmulative_depth, Scalar delta_t, unsigned int size){
      unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;
      while (index < size){
        Scalar small_value = 1e-10;
        Scalar _h = h[index];
        Scalar K_s = hydraulic_conductivity[index];
        Scalar phi_s = capillary_head[index];
        Scalar delta_theta = water_content_diff[index];
        Scalar F_0 = culmulative_depth[index];
        Scalar total_head = phi_s + _h;
        Scalar F_1 = 0.5*(F_0 + delta_t*K_s + sqrt((F_0 + delta_t*K_s)*(F_0 + delta_t*K_s)+4.0*delta_t*K_s*total_head*delta_theta));
        Scalar delta_F = fmin(_h,F_1 - F_0);
        culmulative_depth[index] += delta_F;
        h[index] -= delta_F;
        index += blockDim.x * gridDim.x;
      }

    }


    void cuInfiltrationGreenAmpt(cuFvMappedField<Scalar, on_cell>& h, cuFvMappedField<Scalar, on_cell>& hydraulic_conductivity, cuFvMappedField<Scalar, on_cell>& capillary_head, cuFvMappedField<Scalar, on_cell>& water_content_diff, cuFvMappedField<Scalar, on_cell>& culmulative_depth, Scalar delta_t){

      cuInfiltrationGreenAmptKernel << <BLOCKS_PER_GRID, THREADS_PER_BLOCK >> >(h.data.dev_ptr(), hydraulic_conductivity.data.dev_ptr(), capillary_head.data.dev_ptr(), water_content_diff.data.dev_ptr(), culmulative_depth.data.dev_ptr(), delta_t, h.data.size());

    }


  }

}